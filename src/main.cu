#include "hip/hip_runtime.h"
/**
 * @file main.cu
 * 
 * @author Xujin He (xh1131@nyu.edu)
 * @brief This is cuda code to run
 * @version 0.1
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "debug_printing.h"
#include "linalg_cu.h"

#ifndef __GLOBAL_H
#include "global.h"
#endif

#include <hip/hip_runtime.h>
#include <>

__global__ void computeOffDiagonalL(double* A_values, int* A_columns, int* A_row_ptr, double* D, double* L_values, int* L_columns, int* L_row_ptr, int n);
__global__ void ldlt_colj_cu(int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr, 
                             double *Avalues, int *Acolumns, int *Arow_ptr, double* D, const int col_len, const int row_len);
__global__ void ldlt_Dj_cu(double* D, int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr, int n);
// __global__ void ldlt_cu();

void printCSR(double* L_values, int* L_columns, int* L_row_ptr, int M, int N){
    for (int i = 0; i < M*N; ++i) {
        int valueIndex = L_row_ptr[i];
        for (int j = 0; j < M*N; ++j) {
            if (valueIndex < L_row_ptr[i + 1] && L_columns[valueIndex] == j) {
                std::cout << L_values[valueIndex] << " ";
                ++valueIndex;
            } else {
                std::cout << " 0 ";
            }
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}
void printArray(double *D, int dim) {
    for (int i = 0; i < dim; i++)
    {
        std::cout << D[i] << std::endl;
    }
    
}
const int M = 4;
const int N = 4;
int main(int argc, char const *argv[]){

    const int m = M;
    const int n = N; 
    // const MATRIX_DIM = 1;
    double *u = (double *) malloc(m*n*sizeof(double));
    // loadCSV("../heat_map.csv", u, m*n);
    double *d = (double *) malloc(m*n*sizeof(double));
    double *temp_vec = (double *) malloc(m*n*sizeof(double));

    // Get a 2d Heat Map
    CSRMatrix A = CSRMatrix(m*n, 5*m*n);
    CSRMatrix L = CSRMatrix(m*n, 5*m*m*n); // store the L matrix
    CSRMatrix Lt = CSRMatrix(m*n, 5*m*m*n); // store the L matrix's transpose

    // malloc CSR Matrix A in GPU
    // A_values_d
    // A_columns_d
    // A_row_ptr_d
    double* A_values_d;
    int *A_columns_d, *A_row_ptr_d;
    double *D_d;
    hipMalloc((void **)&A_values_d, 5*m*n*sizeof(double));
    hipMalloc((void **)&A_columns_d, 5*m*n*sizeof(int));
    hipMalloc((void **)&A_row_ptr_d, (m*n+1)*sizeof(int));
    hipMalloc((void **)&D_d, m*n*sizeof(double));
    // init A
    int initA_thread_x = 8;
    int initA_thread_y = 8;
    dim3 initA_grid(m/initA_thread_x+1, n/initA_thread_y+1, 1);
    dim3 initA_block(initA_thread_x, initA_thread_y, 1);
    initBackwardEulerMatrix_kernel<<<initA_grid, initA_block>>>(A_values_d, A_columns_d, A_row_ptr_d, tau*invhsq, m, n);
    hipDeviceSynchronize();
    
    // test init A
    hipMemcpy(A.values, A_values_d, 5*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(A.columns, A_columns_d, 5*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(A.row_ptr, A_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    A.rows = m*n;

    // malloc L and Lt in GPU
    // L_values_d, L_columns_d, L_row_ptr
    // Lt_values_d, Lt_columns_d, Lt_row_ptr
    double *L_values_d, *Lt_values_d;
    int *L_columns_d, *L_row_ptr_d, *Lt_columns_d, *Lt_row_ptr_d;
    hipMalloc((void **)&L_values_d, 5*m*m*n*sizeof(double));
    hipMalloc((void **)&L_columns_d, 5*m*m*n*sizeof(int));
    hipMalloc((void **)&L_row_ptr_d, (m*n+1)*sizeof(int));
    hipMalloc((void **)&Lt_values_d, 5*m*m*n*sizeof(double));
    hipMalloc((void **)&Lt_columns_d, 5*m*m*n*sizeof(int));
    hipMalloc((void **)&Lt_row_ptr_d, (m*n+1)*sizeof(int));
    // init L, LT\t
    int initL_thread = 64;
    dim3 initL_grid(m*n/initL_thread+1, 1, 1);
    dim3 initL_block(initL_thread, 1, 1);
    initL_kernel<<<initL_grid, initL_block>>>(L_values_d, L_columns_d, L_row_ptr_d, m, n);
    hipDeviceSynchronize();
    initLt_kernel<<<initL_grid, initL_block>>>(Lt_values_d, Lt_columns_d, Lt_row_ptr_d, m, n);
    hipDeviceSynchronize();

    // // test init L, Lt
    hipMemcpy(L.values, L_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L.columns, L_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(L.row_ptr, L_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.values, Lt_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.columns, Lt_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.row_ptr, Lt_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    L.rows = m*n;
    Lt.rows = m*n;

    const int grid_size = 1;
    const int block_size = 4;
    for (int J = 0; J < m*n; J++) // loop through all columns
    {
        // kernel update Djj
        ldlt_Dj_cu<<<grid_size,block_size>>>(D_d, J, L_values_d, L_columns_d, L_row_ptr_d, n);
        // kernel update Lij for all i > j
        if (J < n)
        {
            const int col_len = n;
            const int row_len = J;
            ldlt_colj_cu<<<grid_size,block_size>>>(J, L_values_d, L_columns_d, L_row_ptr_d, A_values_d, A_columns_d, A_row_ptr_d, D_d, col_len, row_len); 
        }
        else if (J > m*n-n)
        {
            const int col_len = m*n - J;
            const int row_len = n;
            ldlt_colj_cu<<<grid_size,block_size>>>(J, L_values_d, L_columns_d, L_row_ptr_d, A_values_d, A_columns_d, A_row_ptr_d, D_d, col_len, row_len);
        }
        else
        {
            const int col_len = n;
            const int row_len = n;
            ldlt_colj_cu<<<grid_size,block_size>>>(J, L_values_d, L_columns_d, L_row_ptr_d, A_values_d, A_columns_d, A_row_ptr_d, D_d, col_len, row_len);
        }
        // computeOffDiagonalL<<<grid_size,block_size>>>(J, A_values_d, A_columns_d, A_row_ptr_d, D_d, L_values_d, L_columns_d, L_row_ptr_d, n);
    }
    hipMemcpy(d, D_d, m*n*sizeof(double), hipMemcpyDeviceToHost);
    print_diagonal(d, m*n);
    hipMemcpy(L.values, L_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L.columns, L_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(L.row_ptr, L_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.values, Lt_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.columns, Lt_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.row_ptr, Lt_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    print_csr_matrix(A);
    print_csr_matrix_info(A);
    print_csr_matrix(L);
    print_csr_matrix_info(L);

    // Boundary Condition terms
    double* f_d;
    hipMalloc((void **)&f_d, m*n*sizeof(double));
    int BCthread_x = 8;
    int BCthread_y = 8;
    dim3 BCgrid(m/BCthread_x+1, n/BCthread_y+1, 1);
    dim3 BCblock(BCthread_x, BCthread_y, 1);
    BoundaryCondition_kernel<<<BCgrid, BCblock>>>(f_d, m, n, h);
    hipDeviceSynchronize();

    // Backward Euler steps
    int BEthread = 64;
    int BEblock = m*n/BEthread + 1;
    int total_steps = 1; //endT/tau;
    // allocate memory to store u and b on device
    double *u_d, *b_d;
    hipMalloc((void **)&u_d, m*n*sizeof(double));
    hipMalloc((void **)&b_d, m*n*sizeof(double));
    hipMemcpy(u_d, u, m*n*sizeof(double), hipMemcpyHostToDevice);
    for (int p = 0; p < total_steps; p++)
    {
        // launch kernel to compute updated b
        Updateb_kernel<<<BEblock, BEthread>>>(b_d, u_d, f_d, tau*invhsq, m*n);
        hipDeviceSynchronize();
        // solveAxb(L, Lt, D, b, u, MATRIX_DIM); TBA
    }
    
    hipFree(A_values_d);
    hipFree(A_columns_d);
    hipFree(A_row_ptr_d);
    hipFree(L_values_d);
    hipFree(L_columns_d);
    hipFree(L_row_ptr_d);
    hipFree(Lt_values_d);
    hipFree(Lt_columns_d);
    hipFree(Lt_row_ptr_d);
    hipFree(u_d);
    hipFree(f_d);
    hipFree(b_d);
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }
    std::cout << "reached the end" << std::endl;
}

__device__ double get_ij(int* row_ptr, int* columns, double *values, int i, int j) 
{
    int start = row_ptr[i];
    int end = row_ptr[i + 1];
    for (int k = start; k < end; k++) {
        if (columns[k] == j) {
            return values[k];
        }
    }
    // Column not found, as columns are in ascending order
    return 0;
}
__device__ void set_ij(int* row_ptr, int* columns, double *values, int i, int j, double value) 
{
    int start = row_ptr[i];
    int end = row_ptr[i + 1];
    // Check if the element already exists in the matrix
    for (int k = start; k < end; k++) {
        if (columns[k] == j) {
            values[k] = value;  // Update existing value
            return;
        }
    }
    return;
}

/**
 * @brief 
 * 
 * @param J global column index we are computing
 * @param Lvalues 
 * @param Lcolumns 
 * @param Lrow_ptr 
 * @param Avalues 
 * @param Acolumns 
 * @param Arow_ptr 
 * @param D 
 * @return __global__ 
 */
__global__ void ldlt_colj_cu(int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr, 
                             double *Avalues, int *Acolumns, int *Arow_ptr, double* D, const int col_len, const int row_len)
{    
    // Assume L and Lt have zeros pre-allocated
    // const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // const unsigned int total_t = blockDim.x * gridDim.x;
    
    // assuming total_t << N
    // we let kth thread handle from row k*bh to (k+1)*bh
    // the last thread will handle the middle rows
    // const int col_len = N;            // total number of entries to update ;for starting & interior cols, col length = N; but for J>M*N-N it is (M*N-J) 
    const double DJ = D[J];

    // load shared memory
    // extern __shared__ double Lj_[];
    // const int thread_load = row_len/blockDim.x; 

    // for (int i = 0; i < thread_load; i++)
    // {
    //     Lj_[threadIdx.x*thread_load + i] = Lvalues[Lrow_ptr[J] + threadIdx.x*thread_load + i];
    // }
    // if (threadIdx.x == 0) {
    //     for (int i = thread_load*blockDim.x; i < row_len; i++) Lj_[i] = Lvalues[Lrow_ptr[J] + i];
    // }
    
    __syncthreads();

    for (int local_i = 0; local_i < col_len; local_i++)
    {
        const int global_i = J + 1 + blockIdx.x + gridDim.x * local_i;
        
        if (global_i < J + col_len + 1) 
        {
            const int ith_row_len = row_len - gridDim.x * local_i - blockIdx.x;
            for (int local_k = 0; local_k < ith_row_len; local_k++)
            {
                const int global_k = blockDim.x * local_k + threadIdx.x;
                if (global_k < J) {
                    const int Lik = get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, global_k);
                    const int Ljk = get_ij(Lrow_ptr, Lcolumns, Lvalues, J, global_k);
                    const int Dk = D[global_k];
                    set_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, J, Lik * Ljk * Dk); // sum_{k=0}^{j-1} Lik * Ljk 
                }
            }
            __syncthreads();
            if (threadIdx.x == 0)
            {
                const double Aij = get_ij(Arow_ptr, Acolumns, Avalues, global_i, J);
                const double value = (Aij - get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, J)) / DJ;
                set_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, J, value); // sum_{k=0}^{j-1} Lik * Ljk 
            }  
        }      
    }

    // for (local_i=0; local_i<th; local_i++) // loop the rows, assuming total_t*th >= col_len so that all entries Lijs are covered
    // {
    //     double sumL2 = 0;
    //     const int global_i = J + th*tid + local_i + 1; // computing row global_i, whose value ranges from Lrow_ptr[global_i] to Lrow_ptr[global_i]+N+j-global_i, 
    //     if (global_i < N + 1) 
    //     {
    //         int x = Lrow_ptr[global_i];
    //         // for (int k = 0; k < J; k++)
    //         for (int k = 0; k < J; k++) 
    //         {
    //             // double Lik = Lvalues[x+k]; 
    //             // double Ljk = Lvalues[Lrow_ptr[J]+k];
    //             double Lik = get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, k);
    //             double Ljk = get_ij(Lrow_ptr, Lcolumns, Lvalues, J, k);
    //             sumL2 += Lik * Ljk * D[k];
    //         }
    //         // set Lij = (Aij - sumL2) / Dj, need to figure out if Aij = 0 or not
    //         double Aij = 0;
    //         for (int p = Arow_ptr[global_i]; p < Arow_ptr[global_i+1]; p++)
    //         {
    //             if (J == Acolumns[p]) Aij = Avalues[p];
    //         }
    //         Lvalues[x+J] = (Aij - sumL2) / DJ;
    //     } 
    //     else if (global_i < J + col_len + 1) // middle cols 
    //     {
    //         int x = Lrow_ptr[global_i];
    //         // for (int k = 0; k < J-Lcolumns[x]; k++) 
    //         for (int k = J - N; k < J; k++) 
    //         {
    //             // double Lik = Lvalues[x+k]; 
    //             // double Ljk = Lvalues[Lrow_ptr[J]+k+global_i-J];
    //             double Lik = get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, k);
    //             double Ljk = get_ij(Lrow_ptr, Lcolumns, Lvalues, J, k);
    //             sumL2 += Lik * Ljk * D[k+global_i-J];
    //         }
    //         // set Lij = (Aij - sumL2) / DJ, need to figure out if Aij = 0 or not
    //         double Aij = 0;
    //         for (int p = Arow_ptr[global_i]; p < Arow_ptr[global_i+1]; p++)
    //         {
    //             if (J == Acolumns[p]) Aij = Avalues[p];
    //         }
    //         Lvalues[x+J-global_i+N] = (Aij - sumL2) / DJ;
    //     }
    // }
}

__global__ void ldlt_Dj_cu(double* D, int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < Lrow_ptr[J+1] - Lrow_ptr[J])
    {
        int k = Lrow_ptr[J] + tid;
        double Ljk = Lvalues[k];
        atomicAdd(&D[J], -Ljk * Ljk * D[Lcolumns[k]]); // sum_k Ljk^2 * Dk
    }
    
    __syncthreads();

    if (tid == 0) {
        const double AJJ = 4;
        atomicAdd(&D[J], AJJ);
        Lvalues[Lrow_ptr[J+1]-1] = 1; // Ljj = 1;
    }
}
