#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>

const double h = 0.01;
const double invhsq = 1/h/h;
const double tau = 0.01; // timestep size

int main(int argc, char const *argv[])
{   
    float* Data; // 2D heat map vector = (m*n)
    const int DIM_X = 256; // grid dim = m
    const int DIM_Y = 256; // grid dim = n
    const int DATA_SIZE = DIM_X * DIM_Y * sizeof(float);

    const int MATRIX_DIM = DIM_X * DIM_X * DIM_Y * DIM_Y; // A = (mn x mn), this is extremely large so 

    // randomize a 2d Heat Map
    // srand(time(0));
    // for (i = 0; i < DIM_X*DIM_Y; i++) Data[i] = rand();

    // malloc CSR Matrix A in GPU
    // float* A;
    // cudaMalloc((void **)&A, MATRIX_DIM * sizeof(float));

    return 0;
}