#include "hip/hip_runtime.h"
/**
 * @file main.cu
 * 
 * @author Xujin He (xh1131@nyu.edu)
 * @brief This is cuda code to run
 * @version 0.1
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "debug_printing.h"
#include "linalg_cu.h"

#ifndef __GLOBAL_H
#include "global.h"
#endif

#include <hip/hip_runtime.h>
#include <>

__global__ void ldlt_colj_cu(int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr, 
                             double *Avalues, int *Acolumns, int *Arow_ptr, double* D, const int col_len, const int row_len);
__global__ void ldlt_Dj_cu(double* D, int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr);

const int M = 5;
const int N = 5;

int main(int argc, char const *argv[]){

    const int m = M;
    const int n = N; 
    // const MATRIX_DIM = 1;
    double *u = (double *) malloc(m*n*sizeof(double));
    // loadCSV("../heat_map.csv", u, m*n);
    double *d = (double *) malloc(m*n*sizeof(double));
    double *temp_vec = (double *) malloc(m*n*sizeof(double));

    // Get a 2d Heat Map
    CSRMatrix A = CSRMatrix(m*n, 5*m*n);
    CSRMatrix L = CSRMatrix(m*n, 5*m*m*n); // store the L matrix
    CSRMatrix Lt = CSRMatrix(m*n, 5*m*m*n); // store the L matrix's transpose

    // malloc CSR Matrix A in GPU
    double* A_values_d;
    int *A_columns_d, *A_row_ptr_d;
    double *D_d;
    hipMalloc((void **)&A_values_d, 5*m*n*sizeof(double));
    hipMalloc((void **)&A_columns_d, 5*m*n*sizeof(int));
    hipMalloc((void **)&A_row_ptr_d, (m*n+1)*sizeof(int));
    hipMalloc((void **)&D_d, m*n*sizeof(double));
    // init A
    int initA_thread_x = 8;
    int initA_thread_y = 8;
    dim3 initA_grid(m/initA_thread_x+1, n/initA_thread_y+1, 1);
    dim3 initA_block(initA_thread_x, initA_thread_y, 1);
    initBackwardEulerMatrix_kernel<<<initA_grid, initA_block>>>(A_values_d, A_columns_d, A_row_ptr_d, tau*invhsq, m, n);
    hipDeviceSynchronize();
    
    // test init A
    hipMemcpy(A.values, A_values_d, 5*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(A.columns, A_columns_d, 5*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(A.row_ptr, A_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    A.rows = m*n;

    // malloc L and Lt in GPU
    double *L_values_d, *Lt_values_d;
    int *L_columns_d, *L_row_ptr_d, *Lt_columns_d, *Lt_row_ptr_d;
    hipMalloc((void **)&L_values_d, 5*m*m*n*sizeof(double));
    hipMalloc((void **)&L_columns_d, 5*m*m*n*sizeof(int));
    hipMalloc((void **)&L_row_ptr_d, (m*n+1)*sizeof(int));
    hipMalloc((void **)&Lt_values_d, 5*m*m*n*sizeof(double));
    hipMalloc((void **)&Lt_columns_d, 5*m*m*n*sizeof(int));
    hipMalloc((void **)&Lt_row_ptr_d, (m*n+1)*sizeof(int));
    
    // init L, LT\t
    int initL_thread = 64;
    dim3 initL_grid(m*n/initL_thread+1, 1, 1);
    dim3 initL_block(initL_thread, 1, 1);
    initL_kernel<<<initL_grid, initL_block>>>(L_values_d, L_columns_d, L_row_ptr_d, m, n);
    hipDeviceSynchronize();
    initLt_kernel<<<initL_grid, initL_block>>>(Lt_values_d, Lt_columns_d, Lt_row_ptr_d, m, n);
    hipDeviceSynchronize();


    // // test init L, Lt
    hipMemcpy(L.values, L_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L.columns, L_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(L.row_ptr, L_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.values, Lt_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.columns, Lt_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.row_ptr, Lt_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    L.rows = m*n;
    Lt.rows = m*n;

    const int block_size = 5;
    for (int J = 0; J < m*n; J++) // loop through all columns
    {
        const int row_len = L.row_ptr[J+1] - L.row_ptr[J]; 
        // kernel update Djj
        // compute sumL
        // double Ajj = M_get_ij(A.row_ptr, A.columns, A.values, J, J);
        // Dj_kernel<<<1, J+1>>>(D_d, J,  L_values_d, L_columns_d, L_row_ptr_d, Ajj);
        hipDeviceSynchronize();
        ldlt_Dj_cu<<<1,row_len>>>(D_d, J, L_values_d, L_columns_d, L_row_ptr_d);
        // kernel update Lij for all i > j
        if (J < n)
        {
            const int col_len = n;
            const int row_len = J;
            ldlt_colj_cu<<<col_len,block_size>>>(J, L_values_d, L_columns_d, L_row_ptr_d, A_values_d, A_columns_d, A_row_ptr_d, D_d, col_len, row_len); 
        }
        else if (J > m*n-n)
        {
            const int col_len = m*n - J;
            const int row_len = n;
            ldlt_colj_cu<<<col_len,block_size>>>(J, L_values_d, L_columns_d, L_row_ptr_d, A_values_d, A_columns_d, A_row_ptr_d, D_d, col_len, row_len);
        }
        else
        {
            const int col_len = n;
            const int row_len = n;
            ldlt_colj_cu<<<col_len,block_size>>>(J, L_values_d, L_columns_d, L_row_ptr_d, A_values_d, A_columns_d, A_row_ptr_d, D_d, col_len, row_len);
        }
    }
    hipMemcpy(d, D_d, m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L.values, L_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L.columns, L_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(L.row_ptr, L_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.values, Lt_values_d, 5*m*m*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.columns, Lt_columns_d, 5*m*m*n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Lt.row_ptr, Lt_row_ptr_d, (m*n+1)*sizeof(int), hipMemcpyDeviceToHost);
    print_diagonal(d, m*n);
    // print_csr_matrix(A);
    // print_csr_matrix_info(A);
    print_csr_matrix(L);
    // print_csr_matrix_info(L);

    // Boundary Condition terms
    double* f_d;
    hipMalloc((void **)&f_d, m*n*sizeof(double));
    int BCthread_x = 8;
    int BCthread_y = 8;
    dim3 BCgrid(m/BCthread_x+1, n/BCthread_y+1, 1);
    dim3 BCblock(BCthread_x, BCthread_y, 1);
    BoundaryCondition_kernel<<<BCgrid, BCblock>>>(f_d, m, n, h);
    hipDeviceSynchronize();

    // Backward Euler steps
    int BEthread = 64;
    int BEblock = m*n/BEthread + 1;
    int total_steps = 1; //endT/tau;
    // allocate memory to store u and b on device
    double *u_d, *b_d;
    hipMalloc((void **)&u_d, m*n*sizeof(double));
    hipMalloc((void **)&b_d, m*n*sizeof(double));
    hipMemcpy(u_d, u, m*n*sizeof(double), hipMemcpyHostToDevice);
    for (int p = 0; p < total_steps; p++)
    {
        // launch kernel to compute updated b
        Updateb_kernel<<<BEblock, BEthread>>>(b_d, u_d, f_d, tau*invhsq, m*n);
        hipDeviceSynchronize();
        // solveAxb(L, Lt, D, b, u, MATRIX_DIM); TBA
    }
    
    hipFree(A_values_d);
    hipFree(A_columns_d);
    hipFree(A_row_ptr_d);
    hipFree(L_values_d);
    hipFree(L_columns_d);
    hipFree(L_row_ptr_d);
    hipFree(Lt_values_d);
    hipFree(Lt_columns_d);
    hipFree(Lt_row_ptr_d);
    hipFree(u_d);
    hipFree(f_d);
    hipFree(b_d);
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }
}

__device__ double get_ij(int* row_ptr, int* columns, double *values, const int i, const int j) 
{
    int start = row_ptr[i];
    int end = row_ptr[i + 1];
    for (int k = start; k < end; k++) {
        if (columns[k] == j) {
            return values[k];
        }
    }
    // Column not found, as columns are in ascending order
    return 0;
}
__device__ void set_ij(int* row_ptr, int* columns, double *values, const int i, const int j, double value) 
{
    int start = row_ptr[i];
    int end = row_ptr[i + 1];
    // Check if the element already exists in the matrix
    for (int k = start; k < end; k++) {
        if (columns[k] == j) {
            values[k] = value;  // Update existing value
            return;
        }
    }
    return;
}

#define get_ij_direct(row_ptr, columns, values, i, j) values[row_ptr[i] + j];

/**
 * @brief 
 * 
 * @param J global column index we are computing
 * @param Lvalues 
 * @param Lcolumns 
 * @param Lrow_ptr 
 * @param Avalues 
 * @param Acolumns 
 * @param Arow_ptr 
 * @param D 
 * @return __global__ 
 */
__global__ void ldlt_colj_cu(const int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr, 
                             double *Avalues, int *Acolumns, int *Arow_ptr, double* D, const int col_len, const int row_len)
{    
    // Assume L and Lt have zeros pre-allocated
    // const int col_len = N;            // total number of entries to update ;for starting & interior cols, col length = N; but for J>M*N-N it is (M*N-J) 
    const double DJ = D[J];

    // load shared memory
    extern __shared__ double suml2[];
    suml2[threadIdx.x] = 0;
    __syncthreads();
    
    // gridDim.x = col_len; j + col_length is bottom non-0 element's global row index
    const int global_i = J + 1 + blockIdx.x; 
    int ith_row_len = row_len - 1 - blockIdx.x;
    if (J < N) ith_row_len = row_len;

    for (int local_k = threadIdx.x; local_k < ith_row_len; local_k+=blockDim.x)
    {
        const int global_k = global_i - col_len + local_k;
        // const int kk = (1 + blockIdx.x) + local_k;
        if (global_k >= 0 && global_k < J) {
            const double Lik = get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, global_k);
            const double Ljk = get_ij(Lrow_ptr, Lcolumns, Lvalues, J, global_k);
            // const double Ljk = get_ij_direct(Lrow_ptr, Lcolumns, Lvalues, J, global_k);
            const double Dk = D[global_k];
            suml2[threadIdx.x] += Lik * Ljk * Dk;
        }
    }
    __syncthreads();
    // reduction by sequential addressing
    for (int s = blockDim.x/2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            suml2[threadIdx.x] += suml2[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        const double Aij = get_ij(Arow_ptr, Acolumns, Avalues, global_i, J);
        const double value2 = (Aij - suml2[0]) / DJ;
        set_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, J, value2); // sum_{k=0}^{j-1} Lik * Ljk
        // if (J == M*N-3) {
        //     double x = get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, 12);
        //     double x2 = get_ij(Lrow_ptr, Lcolumns, Lvalues, J, 12); const double xx = x*x2*D[12];
        //     double y = get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, 11);
        //     double y2 = get_ij(Lrow_ptr, Lcolumns, Lvalues, J, 11);const double yy = y*y2*D[11];
        //     double z = get_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, 10);
        //     double z2 = get_ij(Lrow_ptr, Lcolumns, Lvalues, J, 10);const double zz = z*z2*D[10];
        //     double v = (Aij - xx - yy - zz) / DJ;
        //     set_ij(Lrow_ptr, Lcolumns, Lvalues, global_i, J, 114); // sum_{k=0}^{j-1} Lik * Ljk
        // }
    } 
}

__global__ void ldlt_Dj_cu(double* D, int J, double *Lvalues, int *Lcolumns, int *Lrow_ptr)
{
    // load into shared
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int global_k = Lcolumns[Lrow_ptr[J]] + threadIdx.x;

    const double Ljk = Lvalues[Lrow_ptr[J]+tid];
    sdata[tid] = Ljk * Ljk * D[global_k];

    __syncthreads();
    // reduction by sequential addressing
    for (int s = blockDim.x/2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write back
    if (tid == 0) {
        const double AJJ = 4;
        D[J] = AJJ - sdata[0];
        Lvalues[Lrow_ptr[J+1]-1] = 1; // Ljj = 1;
    }
}
